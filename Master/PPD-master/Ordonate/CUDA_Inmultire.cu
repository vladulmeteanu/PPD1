#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""

//inmultirea a doua matrici
__global__ void mat_mul(float *a, float *b, float *c, int N) 
{
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    int col = blockIdx.y*blockDim.y + threadIdx.y;

    if (row < N && col < N) 
    {
        for (int i = 0; i < N; ++i) 
        {
            c[row*N + col] += a[row*N + i] * b[i*N + col];
        }
    }
}

//inmultirea a doi vectori
__global__ void vec_mul(float *a, float *b, float *c, int N) 
{
    int idx = blockDim.x*blockIdx.x+threadIdx.x;

    if (idx < N)
    {
        for (int i = 0; i < N; ++i)
        {
            c[idx] += a[i] * b[idx];
        }
    }
}

//inmultirea unei matrici cu un vectori
__global__ void mat_vec(float *a, float *b, float *c, int N)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int col = blockIdx.y*blockDim.y + threadIdx.y;
	
	if (idx) 
	{
		for(int i=0;i<N;++i)
			c[idx]+=a[idx*N+ i] * b[idx];
	}
} 

int main()
{
    int N = 1024;
    float *a_h, *a_d, *b_h,*b_d,*c_h,*c_d; //declararea matricilor pe host si device
    float *va_h, *va_d, *vb_h, *vb_d, *vc_h, *vc_d; //declararea vectorilor pe host si device
	float *ma_h, *ma_d, *mb_h, *mb_d, *mc_h, *mc_d; //declararea vectorilor pe host si device

//alocarea in memorie pe host si device
    int size = N * N * sizeof(float);
    int v_size = N * sizeof(float);
	int m_size = N * sizeof(float);
	
    a_h = (float*)malloc(size);
    b_h = (float*)malloc(size);
    c_h = (float*)malloc(size);

    va_h = (float*)malloc(v_size);
    vb_h = (float*)malloc(v_size);
    vc_h = (float*)malloc(v_size);
	
	ma_h= (float*)malloc(size);
	mb_h= (float*)malloc(size);
	mc_h= (float*)malloc(size);
	

    hipMalloc((void**)&a_d, size);
    hipMalloc((void**)&b_d, size);
    hipMalloc((void**)&c_d, size);

    hipMalloc((void**)&va_d, v_size);
    hipMalloc((void**)&vb_d, v_size);
    hipMalloc((void**)&vc_d, v_size);
	
	hipMalloc((void**)&ma_d, m_size);
    hipMalloc((void**)&mb_d, m_size);
    hipMalloc((void**)&mc_d, m_size);
	

//initializarea numarului de threaduri si blocks
    dim3 n_threads(32,32,1);
    dim3 n_blocks(32,32,1);
    dim3 v_threads(32, 1, 1);
    dim3 v_blocks(32, 1, 1);
	dim3 m_threads(32,1,1); 
	dim3 m_blocks(32,1,1); //block(N/32,M/32,1) - inmultite valorile de la thread si block trebuie sa dea nr. de pixeli care imi trebuie mie

//popularea vectorilor si matricilor
    for(int i = 0; i < N; ++i) 
    {
        va_h[i] = 1;
        vb_h[i] = 1;
        for (int j = 0; j < N; ++j) 
        {
            a_h[i*N + j] = 1;
            b_h[i*N + j] = 1;
        }
    }

//copierea de pe host pe device a datelor
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
    hipMemcpy(va_d, va_h, v_size, hipMemcpyHostToDevice);
    hipMemcpy(vb_d, vb_h, v_size, hipMemcpyHostToDevice);

//apelarea functiilor de inmultire
    mat_mul << <n_blocks, n_threads >> > (a_d, b_d, c_d, N);
    vec_mul << <v_blocks, v_threads >> > (va_d, vb_d, vc_d, N);
	mat_vec << <m_blocks, m_threads >> > ();
//copierea de pe device pe host a matricii/vectorului rezultant
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);
    hipMemcpy(vc_h, vc_d, v_size, hipMemcpyDeviceToHost);

//afisarea vectorului rezultant
    for (int i = 0; i < N; ++i)
    {
        std::cout << vc_h[i] << " ";
    }

//afisarea matricei rezultante
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            std::cout << c_h[i*N + j] << " ";
        }
        std::cout << std::endl;
    }

//cuda cleanup
    free(a_h);
    free(b_h);
    free(c_h);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
	
    return 0;
}

